#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <wb.h>

#define TILE_WIDTH 16 //do not change this value

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,int numBColumns,int numCRows, int numCColumns) {

  //@@ Insert code to implement basic matrix multiplication here
  //@@ Do not use shared memory to write this kernel

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if ((Row < numCRows) && (column < numCColumns)) {
        float value = 0.0;
        for (int k = 0; k < numAColumns; ++k) {
            value += A[Row * numAColumns + k] * B[k * numBColumns + column];
        }
        C[Row * numCColumns + column] = value;
    }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix

  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  hostC = NULL;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = 0;
  numCColumns = 0;
  if (numAColumns != numBRows) {
      wbLog(TRACE, "numAColumns != numBRows, Break ");
      return 1;
  }
  numCRows = numARows;
  numCColumns = numBColumns;
  unsigned int A_size = numARows * numAColumns * sizeof(float);
  unsigned int B_size = numBRows * numBColumns * sizeof(float);
  unsigned int C_size = numCRows * numCColumns * sizeof(float);
  //@@ Allocate the hostC matrix
  hostC = (float*)malloc(C_size);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void**)&deviceA, A_size);
  hipMalloc((void**)&deviceB, B_size);
  hipMalloc((void**)&deviceC, C_size);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, A_size, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, B_size, hipMemcpyHostToDevice);
  hipMemcpy(deviceC, hostC, C_size, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid((numCColumns - 1) / TILE_WIDTH + 1, (numCRows - 1) / TILE_WIDTH + 1, 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply <<<dimGrid, dimBlock >>> (deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, C_size, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
